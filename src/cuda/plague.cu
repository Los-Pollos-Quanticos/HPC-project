#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hiprand/hiprand_kernel.h>
#include "../utils/utils_cuda.cuh"

__global__ void init_population_kernel(
    int *d_x, int *d_y, int *d_incub, float *d_susc,
    int *d_newInf, int *d_slotIndex,
    hiprandStatePhilox4_32_10_t *states,
    int num_immune, int num_infected, float stddev)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= NP)
    {
        return;
    }

    d_newInf[i] = 0;
    // mark slotIndex invalid as default value
    d_slotIndex[i] = -1;

    if (i < num_immune)
    {
        d_incub[i] = 0;
        d_susc[i] = 0.0f;
    }
    else
    {
        d_incub[i] = (i < num_immune + num_infected) ? (INCUBATION_DAYS + 1) : 0;
        float g = hiprand_normal(&states[i]);
        d_susc[i] = fminf(1.0f, fmaxf(1e-6f, S_AVG + stddev * g));
    }
}

/**
 * Initializes the count, cell slots, and slot index structures
 *
 * @param d_x Array of x coordinates of particles
 * @param d_y Array of y coordinates of particles
 * @param d_cellCount Array to hold the count of particles in each cell
 * @param d_cellSlots Array to hold the slots of particles in each cell
 * @param d_slotIndex Array to hold the index of each particle in its cell
 */
__global__ void buildCellSlots(
    int *d_x, int *d_y, int *d_cellCount,
    int *d_cellSlots, int *d_slotIndex)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= NP)
    {
        return;
    }

    int c = d_x[tid] + d_y[tid] * W;
    if (c < 0)
    {
        d_slotIndex[tid] = -1;
        return;
    }
    // append this particle to cell c
    int pos = atomicAdd(&d_cellCount[c], 1);
    if (pos < MAXP_CELL)
    {
        d_cellSlots[c * MAXP_CELL + pos] = tid;
        d_slotIndex[tid] = pos;
    }
}

// Helpers functions on the device
__device__ bool isImmune(int i, float *d_susc) { return d_susc[i] == 0.0f; }
__device__ bool isDead(int i, int *d_x, int *d_y) { return d_x[i] < 0 || d_y[i] < 0; }
__device__ bool isInfected(int i, int *d_incub) { return d_incub[i] > 0; }

__global__ void infect_kernel(
    int *d_x, int *d_y, int *d_incub, float *d_susc,
    int *d_newInf, int *d_cellCount, int *d_cellSlots)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= NP || !isInfected(tid, d_incub))
    {
        return;
    }

    d_incub[tid]--;

    // Moore neighborhood infection
    int x0 = d_x[tid], y0 = d_y[tid];
    for (int dy = -IRD; dy <= IRD; ++dy)
    {
        int y = y0 + dy;
        if (y < 0 || y >= H)
        {
            continue;
        }
        for (int dx = -IRD; dx <= IRD; ++dx)
        {
            int x = x0 + dx;
            if (x < 0 || x >= W)
            {
                continue;
            }

            // get flat index for cell
            int c = x + y * W;
            // get number of persons in this cell
            int count = d_cellCount[c];
            // recover start of the slots in the d_cellSlots array
            int base = c * MAXP_CELL;
            for (int s = 0; s < count; ++s)
            {
                int i = d_cellSlots[base + s];
                if (i == tid)
                {
                    continue;
                }
                if (!isInfected(i, d_incub) && !isImmune(i, d_susc))
                {
                    float infec = BETA * d_susc[i];
                    if (infec > ITH)
                    {
                        atomicCAS(&d_newInf[i], 0, 1);
                    }
                }
            }
        }
    }
}

__global__ void status_kernel(
    int *d_x, int *d_y, int *d_incub, float *d_susc,
    int *d_newInf, int *d_cellCount,
    int *d_cellSlots, int *d_slotIndex,
    hiprandStatePhilox4_32_10_t *states)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= NP)
        return;

    // if the infected person have completed ints incubation period (we initialize at the beginning with INCUBATION_DAYS + 1)
    if (isInfected(tid, d_incub) && d_incub[tid] == 1)
    {
        float p = hiprand_uniform(&states[tid]);
        bool dies = (p >= MU);
        if (!dies)
        {
            // recover, maybe become immune
            if ((hiprand(&states[tid]) & 1) == 0)
            {
                d_susc[tid] = 0.0f;
            }
            d_incub[tid] = 0;
        }
        else
        {
            // get flat index for cell
            int c = d_x[tid] + d_y[tid] * W;
            // get slot index
            int slot = d_slotIndex[tid];
            // remove form counts of the cell
            int oldCount = atomicSub(&d_cellCount[c], 1);
            // check if I was the last one in the cell
            int last = oldCount - 1;
            // if not the last, swap with the last one
            if (slot != last)
            {
                int other = d_cellSlots[c * MAXP_CELL + last];
                d_cellSlots[c * MAXP_CELL + slot] = other;
                d_slotIndex[other] = slot;
            }
            // mark dead
            d_slotIndex[tid] = -1;
            d_x[tid] = d_y[tid] = -1;
            d_incub[tid] = 0;
        }
    }

    // activate new infections
    if (d_newInf[tid])
    {
        d_incub[tid] = INCUBATION_DAYS + 1;
        d_newInf[tid] = 0;
    }
}

__global__ void move_kernel(
    int *d_x, int *d_y,
    int *d_cellCount, int *d_cellSlots, int *d_slotIndex,
    hiprandStatePhilox4_32_10_t *states)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= NP || isDead(tid, d_x, d_y))
    {
        return;
    }

    unsigned int r1 = hiprand(&states[tid]);
    unsigned int r2 = hiprand(&states[tid]);
    int dx = int(r1 % 3) - 1;
    int dy = int(r2 % 3) - 1;

    int oldX = d_x[tid], oldY = d_y[tid];
    int newX = oldX + dx, newY = oldY + dy;
    if (newX < 0 || newX >= W || newY < 0 || newY >= H)
    {
        return;
    }

    int oldC = oldX + oldY * W;
    int newC = newX + newY * W;
    if (newC == oldC)
    {
        return;
    }

    int oldSlot = d_slotIndex[tid];

    // increments the count of the cell, if the cell was already full rollback, otherwise updates d_cellSlots and d_slotIndex
    int pos = atomicAdd(&d_cellCount[newC], 1);
    if (pos < MAXP_CELL)
    {
        // update d_cellSlots
        d_cellSlots[newC * MAXP_CELL + pos] = tid;
        // update d_slotIndex
        d_slotIndex[tid] = pos;
        // update position
        d_x[tid] = newX;
        d_y[tid] = newY;

        // remove from old cell
        int oldCount = atomicSub(&d_cellCount[oldC], 1);
        // check if I was the last one in the cell
        int last = oldCount - 1;
        // if not the last, swap with the last one
        if (oldSlot != last)
        {
            int other = d_cellSlots[oldC * MAXP_CELL + last];
            d_cellSlots[oldC * MAXP_CELL + oldSlot] = other;
            d_slotIndex[other] = oldSlot;
        }
    }
    else
    {
        // rollback
        atomicSub(&d_cellCount[newC], 1);
    }
}

int main(int argc, char **argv)
{
    bool debug = false;
    for (int i = 1; i < argc; ++i)
        if (!strcmp(argv[i], "--debug"))
            debug = true;

    timespec ts, te;
    timespec ts_rm1, te_rm1;
    clock_gettime(CLOCK_MONOTONIC, &ts);

    // Host coords only use to generate random coordinates
    int *h_x = (int *)malloc(NP * sizeof(int));
    int *h_y = (int *)malloc(NP * sizeof(int));
    gen_random_coords(h_x, h_y);

    // Device allocations
    int *d_x, *d_y, *d_incub, *d_newInf, *d_slotIndex;
    float *d_susc;
    int *d_cellCount, *d_cellSlots;
    hiprandStatePhilox4_32_10_t *d_states;

    hipMalloc(&d_x, NP * sizeof(int));
    hipMalloc(&d_y, NP * sizeof(int));
    hipMalloc(&d_incub, NP * sizeof(int));
    hipMalloc(&d_newInf, NP * sizeof(int));
    hipMalloc(&d_susc, NP * sizeof(float));
    hipMalloc(&d_slotIndex, NP * sizeof(int));

    hipMalloc(&d_cellCount, W * H * sizeof(int));
    hipMalloc(&d_cellSlots, W * H * MAXP_CELL * sizeof(int));
    hipMalloc(&d_states, NP * sizeof(hiprandStatePhilox4_32_10_t));

    int threads = 256;
    int blocks = (NP + threads - 1) / threads;

    init_curand_kernel<<<blocks, threads>>>(d_states, (unsigned long long)time(nullptr));
    clock_gettime(CLOCK_MONOTONIC, &ts_rm1);
    hipMemcpy(d_x, h_x, NP * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, NP * sizeof(int), hipMemcpyHostToDevice);
    clock_gettime(CLOCK_MONOTONIC, &te_rm1);
    free(h_x);
    free(h_y);

    int num_immune = int(NP * IMM);
    int num_infected = int(NP * INFP);
    float stddev = 0.1f;

    init_population_kernel<<<blocks, threads>>>(
        d_x, d_y, d_incub, d_susc, d_newInf, d_slotIndex, d_states,
        num_immune, num_infected, stddev);

    hipMemset(d_cellCount, 0, W * H * sizeof(int));
    buildCellSlots<<<blocks, threads>>>(
        d_x, d_y, d_cellCount,
        d_cellSlots, d_slotIndex);

    if (debug)
        debugState("initial build",
                   d_x, d_y, d_incub, d_susc,
                   d_cellCount, nullptr);

    // Main loop
    for (int day = 0; day < ND; ++day)
    {
        infect_kernel<<<blocks, threads>>>(
            d_x, d_y, d_incub, d_susc, d_newInf,
            d_cellCount, d_cellSlots);
        hipDeviceSynchronize();
        status_kernel<<<blocks, threads>>>(
            d_x, d_y, d_incub, d_susc, d_newInf, d_cellCount,
            d_cellSlots, d_slotIndex, d_states);
        hipDeviceSynchronize();
        move_kernel<<<blocks, threads>>>(
            d_x, d_y, d_cellCount, d_cellSlots, d_slotIndex, d_states);
        hipDeviceSynchronize();

        if (debug)
        {
            char buf[32];
            snprintf(buf, 32, "after day %d", day);
            debugState(buf,
                       d_x, d_y, d_incub, d_susc,
                       d_cellCount, nullptr);
        }
    }

    // Cleanup
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_incub);
    hipFree(d_newInf);
    hipFree(d_susc);
    hipFree(d_slotIndex);
    hipFree(d_cellCount);
    hipFree(d_cellSlots);
    hipFree(d_states);

    clock_gettime(CLOCK_MONOTONIC, &te);
    int runtime = get_time_in_ms(ts, te);
    int runtime_memory_transfer = get_time_in_ms(ts_rm1, te_rm1);
    printf("RunTime: %ld ms\n", get_time_in_ms(ts, te));
    printf("RunTimeEvo: %ld ms\n", runtime - runtime_memory_transfer);
    return 0;
}
