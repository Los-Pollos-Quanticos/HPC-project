#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/tuple.h>
#include <thrust/scatter.h>
#include <thrust/copy.h>
#include <thrust/partition.h>
#include <thrust/binary_search.h>

#include "../utils/utils_cuda.cuh"

__global__ void init_population_kernel(int *d_x, int *d_y, int *d_incub, float *d_susc, int *d_newInf, int *d_cellIdx, hiprandStatePhilox4_32_10_t *states)
{
    float mean = S_AVG;
    float stddev = 0.1f;
    int num_immune = (int)(NP * IMM);
    int num_infected = (int)(NP * INFP);

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= NP)
        return;

    d_cellIdx[i] = d_x[i] + d_y[i] * W;
    d_newInf[i] = 0;

    if (i < num_immune)
    {
        d_incub[i] = 0;
        d_susc[i] = 0.0f;
    }
    else
    {
        d_incub[i] = (i < num_immune + num_infected) ? (INCUBATION_DAYS + 1) : 0;

        float g = hiprand_normal(&states[i]);
        d_susc[i] = fminf(1.0f, fmaxf(1e-6f, mean + stddev * g));
    }
}

struct IsLive
{
    __host__ __device__ bool operator()(const thrust::tuple<int, int, int, float, int, int> &t) const
    {
        return thrust::get<0>(t) >= 0;
    }
};

void rebuildCellMap(
    int *d_cellIdx,
    int *d_x,
    int *d_y,
    float *d_susc,
    int *d_incub,
    int *d_newInf,
    int *d_cellStart,
    int *d_cellCount,
    int *tmpKeys,
    int *tmpCounts)
{
    using thrust::constant_iterator;
    using thrust::device;
    using thrust::device_ptr;
    using thrust::make_tuple;
    using thrust::make_zip_iterator;

    // 1) pack values into a zip‐iterator
    auto zip_begin = make_zip_iterator(
        make_tuple(
            device_ptr<int>(d_cellIdx),
            device_ptr<int>(d_x),
            device_ptr<int>(d_y),
            device_ptr<float>(d_susc),
            device_ptr<int>(d_incub),
            device_ptr<int>(d_newInf)));
    auto zip_end = zip_begin + NP;

    // 2) partition out the “dead” cells
    auto live_end = thrust::stable_partition(device, zip_begin, zip_end, IsLive());
    int live_count = live_end - zip_begin;

    // 3) sort live cells by their cellIdx (keys are in-place in d_cellIdx[]; values are zipped)
    thrust::sort_by_key(
        device,
        d_cellIdx,
        d_cellIdx + live_count,
        make_zip_iterator(make_tuple(d_x, d_y, d_susc, d_incub, d_newInf)));

    // 4) run‐length encode (i.e. unique keys + counts)
    //    → writes unique keys into tmpKeys[0..numUnique)
    //    → writes counts    into tmpCounts[0..numUnique)
    int *endKeyPtr;
    int *endCountPtr;
    thrust::tie(endKeyPtr, endCountPtr) = thrust::reduce_by_key(
        device,
        d_cellIdx, d_cellIdx + live_count,
        constant_iterator<int>(1),
        tmpKeys,
        tmpCounts);
    int numUnique = endKeyPtr - tmpKeys;

    // 5) exclusive‐scan counts → directly into d_cellStart[key]
    thrust::exclusive_scan(
        device,
        tmpCounts, tmpCounts + numUnique,
        thrust::make_permutation_iterator(
            device_ptr<int>(d_cellStart),
            device_ptr<int>(tmpKeys)),
        0);

    // 6) scatter counts → directly into d_cellCount[key]
    thrust::scatter(
        device,
        tmpCounts, tmpCounts + numUnique,
        tmpKeys,
        d_cellCount);
}

__device__ bool isImmune(int i, float *d_susc)
{
    return d_susc[i] == 0.0f;
}

__device__ bool isDead(int i, int *d_cellIdx)
{
    return d_cellIdx[i] < 0;
}

__device__ bool isInfected(int i, int *d_incub)
{
    return d_incub[i] > 0;
}

__global__ void infect_kernel(
    int *d_x,
    int *d_y,
    int *d_incub,
    float *d_susc,
    int *d_newInf,
    int *d_cellStart,
    int *d_cellCount)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= NP)
        return;

    if (!isInfected(tid, d_incub))
        return;

    d_incub[tid]--;

    int x0 = d_x[tid], y0 = d_y[tid];

    for (int dy = -IRD; dy <= IRD; ++dy)
    {
        int y = y0 + dy;
        if (y < 0 || y >= H)
            continue;
        for (int dx = -IRD; dx <= IRD; ++dx)
        {
            int x = x0 + dx;
            if (x < 0 || x >= W)
                continue;

            int c = x + y * W;
            int start = d_cellStart[c];
            int count = d_cellCount[c];
            for (int i = start; i < start + count; ++i)
            {
                if (i == tid)
                    continue;

                if (!isInfected(i, d_incub) && !isImmune(i, d_susc))
                {
                    float infec = BETA * d_susc[i];
                    if (infec > ITH)
                    {
                        // If d_newInf[i] is 0, it sets it to 1 and returns 0 → this thread succeeds
                        atomicCAS(&d_newInf[i], 0, 1);
                    }
                }
            }
        }
    }
}

__global__ void status_kernel(
    int *d_x,
    int *d_y,
    int *d_incub,
    float *d_susc,
    int *d_newInf,
    int *d_cellIdx,
    int *d_cellCount,
    hiprandStatePhilox4_32_10_t *d_curandStates)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= NP)
        return;

    if (isInfected(tid, d_incub) && d_incub[tid] == 1)
    {
        float p = hiprand_uniform(&d_curandStates[tid]);
        if (p < MU)
        {
            // recover
            if ((hiprand(&d_curandStates[tid]) & 1) == 0)
            {
                d_susc[tid] = 0.0f;
            }

            d_incub[tid] = 0;
        }
        else
        {
            int cellPos = d_cellIdx[tid];
            if (cellPos >= 0)
            {
                atomicSub(&d_cellCount[cellPos], 1);
            }
            d_cellIdx[tid] = -1;
            d_x[tid] = -1;
            d_y[tid] = -1;
            d_incub[tid] = 0;
        }
    }

    if (d_newInf[tid])
    {
        d_incub[tid] = INCUBATION_DAYS + 1;
        d_newInf[tid] = 0;
    }
}

__global__ void move_kernel(
    int *d_x,
    int *d_y,
    int *d_cellIdx,
    int *d_cellCount,
    hiprandStatePhilox4_32_10_t *states)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= NP)
        return;

    if (isDead(tid, d_cellIdx))
        return;

    unsigned int r1 = hiprand(&states[tid]);
    unsigned int r2 = hiprand(&states[tid]);
    int dx = (int)(r1 % 3) - 1;
    int dy = (int)(r2 % 3) - 1;

    int oldX = d_x[tid], oldY = d_y[tid];
    int newX = oldX + dx, newY = oldY + dy;

    if (newX < 0 || newX >= W || newY < 0 || newY >= H)
        return;
    int oldCell = oldX + oldY * W;
    int newCell = newX + newY * W;
    if (newCell == oldCell)
        return;

    int prev = atomicAdd(&d_cellCount[newCell], 1);
    if (prev >= MAXP_CELL)
    {
        atomicSub(&d_cellCount[newCell], 1);
        return;
    }

    atomicSub(&d_cellCount[oldCell], 1);

    d_x[tid] = newX;
    d_y[tid] = newY;
    d_cellIdx[tid] = newCell;
}

int main(int argc, char **argv)
{
    bool debugEnabled = false;
    for (int i = 1; i < argc; ++i)
        if (strcmp(argv[i], "--debug") == 0)
            debugEnabled = true;

    printf("Simulation started\n");
    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC, &start);

    log_memory_usage("Start");

    int *h_x = (int *)malloc(NP * sizeof(int));
    int *h_y = (int *)malloc(NP * sizeof(int));

    int *d_x;
    int *d_y;
    int *d_incub;
    float *d_susc;
    int *d_newInf;
    hipMalloc(&d_x, NP * sizeof(int));
    hipMalloc(&d_y, NP * sizeof(int));
    hipMalloc(&d_incub, NP * sizeof(int));
    hipMalloc(&d_susc, NP * sizeof(float));
    hipMalloc(&d_newInf, NP * sizeof(int));

    int *d_cellIdx;
    int *d_cellStart;
    int *d_cellCount;
    hipMalloc(&d_cellIdx, NP * sizeof(int));
    hipMalloc(&d_cellStart, W * H * sizeof(int));
    hipMalloc(&d_cellCount, W * H * sizeof(int));

    int *tmpKeys;
    int *tmpCounts;
    hipMalloc(&tmpKeys, NP * sizeof(int));
    hipMalloc(&tmpCounts, NP * sizeof(int));

    int threads = 256;
    int blocks = (NP + threads - 1) / threads;

    hiprandStatePhilox4_32_10_t *d_curandStates;
    hipMalloc(&d_curandStates, NP * sizeof(hiprandStatePhilox4_32_10_t));
    init_curand_kernel<<<blocks, threads>>>(d_curandStates, (unsigned long long)time(NULL));

    gen_random_coords(h_x, h_y);
    hipMemcpy(d_x, h_x, NP * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, NP * sizeof(int), hipMemcpyHostToDevice);
    free(h_x);
    free(h_y);

    init_population_kernel<<<blocks, threads>>>(d_x, d_y, d_incub, d_susc, d_newInf, d_cellIdx, d_curandStates);

    log_memory_usage("After population init");

    rebuildCellMap(d_cellIdx, d_x, d_y, d_susc, d_incub, d_newInf, d_cellStart, d_cellCount, tmpKeys, tmpCounts);

    log_memory_usage("After rebuildCellMap");

    if (debugEnabled)
    {
        debugState("after rebuildCellMap",
                   d_x, d_y, d_cellIdx, d_incub, d_susc, d_cellCount,
                   d_cellStart);
    }

    for (int day = 0; day < ND; ++day)
    {
        infect_kernel<<<blocks, threads>>>(d_x, d_y, d_incub, d_susc, d_newInf, d_cellStart, d_cellCount);
        status_kernel<<<blocks, threads>>>(d_x, d_y, d_incub, d_susc, d_newInf, d_cellIdx, d_cellCount, d_curandStates);
        move_kernel<<<blocks, threads>>>(d_x, d_y, d_cellIdx, d_cellCount, d_curandStates);
        rebuildCellMap(d_cellIdx, d_x, d_y, d_susc, d_incub, d_newInf, d_cellStart, d_cellCount, tmpKeys, tmpCounts);
        log_memory_usage("After day");

        if (debugEnabled)
        {
            char label[32];
            snprintf(label, sizeof(label), "after day %d", day);
            debugState(label,
                       d_x, d_y, d_cellIdx, d_incub, d_susc, d_cellCount,
                       d_cellStart);
        }

        hipDeviceSynchronize();
    }

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_susc);
    hipFree(d_incub);
    hipFree(d_newInf);
    hipFree(d_cellIdx);
    hipFree(d_cellStart);
    hipFree(d_cellCount);
    hipFree(d_curandStates);

    log_memory_usage("After cleanup");

    clock_gettime(CLOCK_MONOTONIC, &end);
    printf("Time: %ld ms\n", get_time_in_ms(start, end));
}